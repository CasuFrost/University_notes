#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vecAddKernel(float *A, float *B, float *C, int n)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n)
    {
        C[index] = A[index] + B[index];
    }
}

void vecAdd(float *h_A, float *h_B, float *h_C, int n)
{
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, n * sizeof(float));
    hipMemcpy(d_A, h_A, n * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void **)&d_B, n * sizeof(float));
    hipMemcpy(d_B, h_B, n * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void **)&d_C, n * sizeof(float));

    vecAddKernel<<<ceil(n / 256.0), 256>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();
    hipMemcpy(h_C, d_C, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

#define SIZE 10000

int main()
{
    float a[SIZE];
    float b[SIZE];
    float c[SIZE];
    for (size_t i = 0; i < SIZE; i++)
    {
        a[i] = 1.5;
        b[i] = 0.5;
    }
    vecAdd(a, b, c, SIZE);
}